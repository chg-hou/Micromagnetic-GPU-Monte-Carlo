#include "hip/hip_runtime.h"
/*
 * link  :     cufft cublas hiprand boost_system boost_filesystem
 *               // must link boost_system before boost_filesystem    http://stackoverflow.com/questions/9723793/undefined-reference-to-boostsystemsystem-category-when-compiling
 * 				-lboost_system -lboost_filesystem
 * include: /usr/local/cuda-8.0/samples/common/inc/
 * c++11:   -std=c++11 -stdlib=libc++     //needed by boost     //http://stackoverflow.com/questions/19469887/segmentation-fault-with-boostfilesystem
 *
 * -I/usr/local/cuda-8.0/samples/common/inc/ -G -g -O0 -std=c++11
 *
 * environment COPTS   = '-g -O0'  to disable optim
 */


/*
 *    /home/cg/cuda-workspace/abc.ini -gpu=0  -tensorcache="/home/cg/__tensor_cache/"
 *    abc.ini -gpu=0  -tensorcache="/home/cg/__tensor_cache/"
 */

// windows copy along with boost_system-vc140-mt-1_62
// include path :     E:\local\boost_1_62_0\         C:\ProgramData\NVIDIA Corporation\CUDA Samples\v8.0\common\inc
//					compute_60,sm_60
// linker:  addtion path :  E:\local\boost_1_62_0\lib64-msvc-14.0;%(AdditionalLibraryDirectories)
// input cufft.lib;cublas.lib;hiprand.lib;boost_system-vc140-mt-1_62.lib;boost_filesystem-vc140-mt-1_62.lib;%(AdditionalDependencies)
// TODO: cufft plan many
#include "demag_cu.h"
#include "constant.h"
#include "effective_H.h"
#include "mc_sampling.h"
#include "mx_init.h"
#include "ovf_io.h"
#include "terminal_print.h"
#include "parse_input_ini.h"
#include "GPU_selector.h"
#include "demag_mirror.h"

#include ""


#include <boost/filesystem.hpp>
#include <boost/filesystem/path.hpp>
#include <boost/filesystem/operations.hpp>
#include <boost/progress.hpp>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <ctime>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <hipfft/hipfft.h>
#include <fstream>

#include <string>


#include <chrono>
#include <thread>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

#ifdef _WIN32
        
#else
     #include <unistd.h>
#endif

#ifdef DEBUG_H_FIELD
	#include "mc_sampling_debug.h"
#endif

#include <signal.h>
#include <stdlib.h>

#include <timer.h>


bool CTRL_C_QUIT_FLAG = false;
void ctrl_c_handler(int s)
{
	printf("Caught signal %d\n",s);
	CTRL_C_QUIT_FLAG = true;
	return;
}

int main(int argc, char *argv[]) {



	//================================
	using namespace std;
	setbuf(stdout,NULL);
	if (argc==1)
	{
		cout<<"need input file. \n";
		cout<<"arg format:  input_cfg_file.ini  -gpu=0   -tensorcache=\"/mnt/cachepath/\" \n";
		return 0;
	}

	string input_cfg_filename = getCfgFilename( argc,(const char **) argv);
	getCudaDevice( argc, (const char **) argv);

//=========================init para=======
	int  nx; int  ny; int  nz;
	double  dx;double  dy;double  dz;

	FLOAT_   Bextx;FLOAT_   Bexty;FLOAT_   Bextz;

	int  pbc_x; int  pbc_y; int  pbc_z;

	FLOAT_   ms; FLOAT_   Aex;FLOAT_   Dind;

	FLOAT_   anisUx; FLOAT_   anisUy;FLOAT_   anisUz;FLOAT_   Ku1;

	bool   use_random_init;
	long long   randomseed;
	string   ovf_filename;

	FLOAT_   Temperature_start; FLOAT_   Temperature_end;
	FLOAT_   Temperature_step;
	bool Temperature_use_exp;

	long   circle_per_stage ;
	long   terminal_output_period;
	long   energy_output_period;
	long   ms_output_period;

	int   rand_block_size;
	bool cal_demag_flag;

	//-----------------------------

	//============catch ctrl-c=================
	signal(SIGINT, &ctrl_c_handler);


	//----------------------------------------




	boost::filesystem::path input_cfg_filename_path(input_cfg_filename);
	if (!boost::filesystem::exists(input_cfg_filename_path))
		input_cfg_filename_path = boost::filesystem::current_path() / input_cfg_filename_path ;
	input_cfg_filename_path = boost::filesystem::absolute(input_cfg_filename_path);
	std::cout << "input_cfg_path is : "<<input_cfg_filename_path<<endl;
	input_cfg_filename = input_cfg_filename_path.string();

	if (!parse_input_ini(input_cfg_filename,
			  nx,   ny,   nz,
			  dx,  dy,  dz,
			   Bextx,   Bexty,   Bextz,
			  pbc_x,   pbc_y,   pbc_z,
			   ms,    Aex,   Dind,
			   anisUx,    anisUy,   anisUz,   Ku1,
			  use_random_init,
			    randomseed,
			ovf_filename,
			   Temperature_start,    Temperature_end,
			   Temperature_step,
			   circle_per_stage ,
			   terminal_output_period,
			   energy_output_period,
			   ms_output_period,
			   rand_block_size,
			   cal_demag_flag,
			   Temperature_use_exp))
		return 0;
	cout_input_ini(
			  nx,   ny,   nz,
			  dx,  dy,  dz,
			   Bextx,   Bexty,   Bextz,
			  pbc_x,   pbc_y,   pbc_z,
			   ms,    Aex,   Dind,
			   anisUx,    anisUy,   anisUz,   Ku1,
			   use_random_init,
			    randomseed,
			   ovf_filename,
			   Temperature_start,    Temperature_end,
			   Temperature_step,
			   circle_per_stage ,
			   terminal_output_period,
			   energy_output_period,
			   ms_output_period,
			   rand_block_size,
			   cal_demag_flag,
			   Temperature_use_exp);

	boost::filesystem::path output_path =
			(input_cfg_filename_path.parent_path()/input_cfg_filename_path.stem()).string()
			+ ".output";	;//boost::filesystem::path(".\\energy");
	if (! boost::filesystem::exists(output_path) )
	{
		if(boost::filesystem::create_directories(output_path)) {
				std::cout << "Successfully create output folder: "<< output_path << "\n";
			}
		else
		{
			std::cout << "Fail to create output folder: "<< output_path << "\n";
			return 0;
		}
	}else
	{
		namespace fs = boost::filesystem;
		fs::directory_iterator end_iter;
		for (fs::directory_iterator dir_itr(output_path);
		          dir_itr != end_iter;
		          ++dir_itr)
		    {
		      try
		      {
		    	  std::string extension = dir_itr->path().extension().string();
		    	  if (extension==".ovf" || extension==".png"|| extension==".jpg"
		    			  || extension==".jpeg")
		    	  {
		    		  fs::remove(dir_itr->path());
		    	  }

		      }
		      catch (const std::exception & ex)
		      {
		        std::cout << dir_itr->path().filename() << " " << ex.what() << std::endl;
		      }
		    }

	}
	cout<<"output_path: "<<output_path<<endl;

	string energy_filename_string = (output_path  / boost::filesystem::path("energy.txt")).string();//output_path ;// / boost::filesystem::path("energy.txt"); //output_path.string() ;
	cout<<"energy_filename_string: "<<energy_filename_string<<endl;
	{
		FILE * fp;
		fp = fopen(energy_filename_string.c_str(),"w");
		fclose(fp);
	}


	FLOAT_ Temperature = Temperature_start;

	FLOAT_ mu0 = 4e-7 * CONST_PI;
	FLOAT_ factor_H_exch = 2.0 * Aex / mu0 / ms;
	if (randomseed<0)
	{
		randomseed = time(NULL);
		cout<<"Use random seed: "<<randomseed<<endl;
	}



	bool padding_x = (pbc_x == 0);
	bool padding_y = (pbc_y == 0);
	bool padding_z = (pbc_z == 0);

	int nx_padded = padding_x ? (2 * nx - 1) : nx;
	int ny_padded = padding_y ? (2 * ny - 1) : ny;
	int nz_padded = padding_z ? (2 * nz - 1) : nz;

	long meshsize = nx * ny * nz;
	long meshsize_padded = nx_padded * ny_padded * nz_padded;

	//===============
	ovf_io::OvfInfo ovfinfo;
	ovfinfo.xnodes = nx;
	ovfinfo.ynodes = ny;
	ovfinfo.znodes = nz;
	ovfinfo.xstepsize = dx;
	ovfinfo.ystepsize = dy;
	ovfinfo.zstepsize = dz;
	//===============
	//const int blocksize = 10;
	int blockDim_x = 5;
	int blockDim_y = 5;
	int blockDim_z = 1;

	dim3 dimBlock(blockDim_x, blockDim_y, blockDim_z);
	dim3 dimGrid((nx_padded + blockDim_x - 1) / blockDim_x,
			(ny_padded + blockDim_y - 1) / blockDim_y,
			(nz_padded + blockDim_z - 1) / blockDim_z);

	dim3 dimBlock_nopadding(blockDim_x, blockDim_y, blockDim_z);
	dim3 dimGrid_nopadding((nx + blockDim_x - 1) / blockDim_x,
			(ny + blockDim_y - 1) / blockDim_y,
			(nz + blockDim_z - 1) / blockDim_z);

	/*==========================load Mx ===========================*/
	FLOAT_ * h_Mx_padded;
	FLOAT_ * h_My_padded;
	FLOAT_ * h_Mz_padded;
	{
		//
//		double h_Mx[meshsize];
//		double h_My[meshsize];
//		double h_Mz[meshsize];
		double * h_Mx = new double[meshsize];
		double * h_My = new double[meshsize];
		double * h_Mz = new double[meshsize];

#ifdef USE_ONE_HOT_MX_INIT
		for(int i=0;i<meshsize;i++)
		{
			h_Mx[i]=0;
			h_My[i]=0;
			h_Mz[i]=0;
		}
		h_Mx[0]=1;
#else
		if (use_random_init)
		{
			//randInitMx(T * Mx, T * My, T * Mz, int nx, int ny, int nz, unsigned long long seed = 1234L )
			randInitMx( h_Mx,  h_My,  h_Mz,  nx, ny, nz, randomseed);

		}
		else
		{
			if (!boost::filesystem::exists(boost::filesystem::path(ovf_filename)))
				{cout<<"OVF file not found: "<<ovf_filename<<endl;return 0;}
			if (!(ovf_io::parseOvfFile<double>(ovf_filename, ovfinfo, h_Mx,  h_My,  h_Mz)))
				{cout<<"Error, quit now.";return 0;}
		}
#endif
		h_Mx_padded = new FLOAT_[meshsize_padded];
		h_My_padded = new FLOAT_[meshsize_padded];
		h_Mz_padded = new FLOAT_[meshsize_padded];

		copyMxtoMxpadded<double,FLOAT_>(h_Mx, h_My, h_Mz,  nx,  ny,  nz,
				h_Mx_padded, h_My_padded, h_Mz_padded, nx_padded, ny_padded, nz_padded);
		delete h_Mx,h_My,h_Mz;
	}

	/*--------------------------load Mx ---------------------------*/



	/*==========================K tensor for pbc=========*/
	double Kxx_mirror, Kyy_mirror, Kzz_mirror;
	double Kxy_mirror, Kxz_mirror, Kyz_mirror;
	demag_mirror::calDemagMirrorTensor( nx_padded,  ny_padded,  nz_padded,
			 pbc_x,  pbc_y,  pbc_z,
			 dx, dy ,  dz,
			 Kxx_mirror, Kxy_mirror, Kxz_mirror,
			 Kyy_mirror, Kyz_mirror, Kzz_mirror
	);


	/*-----------------------K tensor for pbc-----------------------*/

	/*=======================init random block================*/
	hiprandGenerator_t randgen;
	FLOAT_ * dRandPool;
	FLOAT_ * dRandPool_reject;
	int randPoolCount = rand_block_size;
	//cout<<sizeof(FLOAT_) * 3 * meshsize *  rand_block_size ;
	checkCudaErrors(
			hipMalloc((void ** )&dRandPool, sizeof(FLOAT_) * 3 * meshsize *  rand_block_size ));
	checkCudaErrors(
			hipMalloc((void ** )&dRandPool_reject, sizeof(FLOAT_) * meshsize *  rand_block_size ));
	CURAND_CALL(hiprandCreateGenerator(&randgen, HIPRAND_RNG_PSEUDO_DEFAULT));
//	HIPRAND_RNG_PSEUDO_DEFAULT
//	HIPRAND_RNG_PSEUDO_XORWOW
//	HIPRAND_RNG_PSEUDO_MRG32K3A
//	HIPRAND_RNG_PSEUDO_MTGP32
//	HIPRAND_RNG_PSEUDO_MT19937
//	HIPRAND_RNG_PSEUDO_PHILOX4_32_10
//	HIPRAND_RNG_QUASI_DEFAULT
//	HIPRAND_RNG_QUASI_SOBOL32
//	HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32
//	HIPRAND_RNG_QUASI_SOBOL64
//	HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64

	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(randgen, randomseed));
//	 /* Generate n floats on device */
//#ifdef USE_DOULBE_IN_GPU
//	CURAND_CALL(hiprandGenerateNormalDouble(randgen, dRandPool, 3 * meshsize * rand_block_size,
//				0.0,1.0));	//double mean, double stddev
//	CURAND_CALL(hiprandGenerateUniformDouble(randgen, dRandPool_reject,  meshsize *  rand_block_size));
//#else
//	CURAND_CALL(hiprandGenerateNormal(randgen, dRandPool, 3 * meshsize * rand_block_size,
//			0.0f,1.0f));	//float mean, float stddev
//	// excluding 0.0 and including 1.0      ( 0.0 , 1.0]
//	CURAND_CALL(hiprandGenerateUniform(randgen, dRandPool_reject, meshsize * rand_block_size));
//#endif
//	/*------------ init random block           */


	/* ================================init Kxx fft==================================*/
	cout << endl << "Dimension: nx ny nz meshsize";
	cout << endl << nx << " " << ny << " " << nz <<" "<<meshsize<< endl;
	cout << "nx_padded ny_padded nz_padded meshsize_padded";
	cout << endl << nx_padded << " " << ny_padded << " " << nz_padded <<" "<<meshsize_padded<< endl;

	cout<<"Calculating demag tensor...\n";
	double * Kxx, *Kxy, *Kxz, *Kyy, *Kyz, *Kzz;

	checkCudaErrors(
			hipMalloc((void ** )&Kxx, sizeof(double) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kxy, sizeof(double) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kxz, sizeof(double) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kyy, sizeof(double) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kyz, sizeof(double) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kzz, sizeof(double) * meshsize_padded));






//	ktensorKernel<<<dimGrid, dimBlock>>>(nx, ny, nz, dx, dy, dz, Kxx, Kxy, Kxz,
//			Kyy, Kyz, Kzz);

	string tensor_cache_filename;
	{
		char filename_buff[200];
		snprintf(filename_buff,200,"cache_%d_%d_%d-%g_%g_%g-%d_%d_%d.ovf",nx_padded, ny_padded, nz_padded, dx,
				dy, dz, pbc_x, pbc_y, pbc_z);
		tensor_cache_filename = filename_buff;
	}
	bool succefully_load_tensor_cache = false;
	if(boost::filesystem::exists(boost::filesystem::path(tensor_cache_path) / boost::filesystem::path(tensor_cache_filename) ))
	{
		//load
		cout<<"  loading cached tensor... "<<boost::filesystem::path(tensor_cache_path) / boost::filesystem::path(tensor_cache_filename) ;
		ifstream ifs( (boost::filesystem::path(tensor_cache_path) / boost::filesystem::path(tensor_cache_filename)).string(),
				ios::binary);
		if (ifs)
		{
			double * h_K_tmp = new double[meshsize_padded];
			try
			{

				double * d_K_array[6]={Kxx, Kxy, Kxz, Kyy, Kyz, Kzz};
				for (int i = 0; i <6; i++)
				{
					ifs.read(reinterpret_cast<char *>(h_K_tmp), sizeof(double) * meshsize_padded );
					checkCudaErrors(hipMemcpy(d_K_array[i], h_K_tmp, sizeof(double) * meshsize_padded,
													hipMemcpyHostToDevice));
				}
				succefully_load_tensor_cache = true;
				cout<<"  Done.\n";

			}
			catch(const std::exception &e)
			{succefully_load_tensor_cache = false;cout<<"  ERROR.\n";};
			delete h_K_tmp;
		}
		ifs.close();
	}
	if (!succefully_load_tensor_cache)
	{
		//cal
		{
			cout<<"		Calculating in real space ...";

#ifdef USE_CPU_TO_CALCULATE_K_TENSOR
			ktensorcpu::ktensorCPU(nx_padded, ny_padded, nz_padded, dx,
					dy, dz, pbc_x, pbc_y, pbc_z, Kxx, Kxy, Kxz, Kyy, Kyz, Kzz,CTRL_C_QUIT_FLAG);
#else
			ktensorKernel2_step0_init<<<dimGrid, dimBlock>>>(nx_padded, ny_padded, nz_padded,
					Kxx, Kxy, Kxz, Kyy, Kyz, Kzz);

			{
				printf("....................");

//				float count = 0;
				long total = (2*max(pbc_x,0)+1)*(2*max(pbc_y,0)+1)*(2*max(pbc_z,0)+1);
				boost::progress_display show_progress( total );
				for (int pbc_idx_x = -max(pbc_x,0) ; pbc_idx_x <= max(pbc_x,0); pbc_idx_x++) {
					for (int pbc_idx_y = -max(pbc_y,0); pbc_idx_y <= max(pbc_y,0); pbc_idx_y++) {
						for (int pbc_idx_z = -max(pbc_z,0); pbc_idx_z <= max(pbc_z,0); pbc_idx_z++) {

							if (CTRL_C_QUIT_FLAG) break;
//							printf("\b\b\b\b\b\b\b\b\b\b\b\b...% 3.1f %% ...",(count++)*100.0/total);

	//						ktensorKernel<<<dimGrid, dimBlock>>>(nx_padded, ny_padded, nz_padded, dx,
	//											dy, dz, pbc_x, pbc_y, pbc_z, Kxx, Kxy, Kxz, Kyy, Kyz, Kzz, d_data);

							ktensorKernel2_step1<<<dimGrid, dimBlock>>>(nx_padded, ny_padded, nz_padded, dx,
									dy, dz,  Kxx, Kxy, Kxz, Kyy, Kyz, Kzz,
									pbc_idx_x, pbc_idx_y, pbc_idx_z);
							++show_progress;
							hipDeviceSynchronize();
						}
					}
				}
//				printf("\b\b\b\b\b\b\b\b\b\b\b\b...100%% ...\n");
//				cout<<count<<"  "<<total<<endl;
				printf("   Done.\n");
			}
			ktensorKernel2_step2_div_const<<<dimGrid, dimBlock>>>(nx_padded, ny_padded, nz_padded,
					dx, dy, dz,
					Kxx, Kxy, Kxz, Kyy, Kyz, Kzz);

//			volatile int * d_data, * h_data;
//			hipHostAlloc((void **)& h_data,sizeof(int), hipHostMallocMapped);
//			hipHostGetDevicePointer((int **) &d_data, (int *) h_data,0);
//			* h_data = 0;
//
////			hipEvent_t start,stop;
////			hipEventCreate(&start);hipEventCreate(&stop);
////			hipEventRecord(start);
//
//			ktensorKernel<<<dimGrid, dimBlock>>>(nx_padded, ny_padded, nz_padded, dx,
//					dy, dz, pbc_x, pbc_y, pbc_z, Kxx, Kxy, Kxz, Kyy, Kyz, Kzz, d_data);
//
//
////			hipEventRecord(stop);
//
//			unsigned int num_blocks = dimGrid.x * dimGrid.y;
//			float my_progress = 0.0f;
//
//	#ifndef  _WIN32
//				do{
//					//boost::this_thread::sleep(boost::posix_time::milliseconds(100));
//					std::this_thread::sleep_for(std::chrono::microseconds(100000));
//					//usleep(100000);
//					int value1 = *h_data;
//					float kern_progress = (float)value1 /(float) num_blocks * ((2*max(pbc_x,0)+1)* (2*max(pbc_y,0)+1) * (2*max(pbc_z,1)+1));
//					if((kern_progress-my_progress)>0.05f)
//					{
//						printf("...%3.1f %% ...\n", kern_progress*100.0);
//
//						my_progress= kern_progress;
//					}
//				}while (my_progress<0.99f);
//	#endif
#endif
			cout<<"Done.\n";

			if (pbc_x<0) pbc_x=1;
			if (pbc_y<0) pbc_y=1;
			if (pbc_z<0) pbc_z=1;
			printf("pbc_x = %d, pbc_y = %d, pbc_z = %d\n",pbc_x,pbc_y,pbc_z);
//			hipEventSynchronize(stop);
//			float et;
//			hipEventElapsedTime(&et, start, stop);
//			hipDeviceSynchronize();
//			printf("Elaspsed time = %f ms", et);
		}
		//save
		if(boost::filesystem::exists(boost::filesystem::path(tensor_cache_path) ))
		{
			string cache_filename = (boost::filesystem::path(tensor_cache_path) / boost::filesystem::path(tensor_cache_filename)).string();
			cout<<"Write tensor to cache: "<<cache_filename<<endl;
			ofstream ofs( cache_filename,
							ios::binary);
			if (ofs)
			{
				try
				{
					double *  h_K_tmp = new double[meshsize_padded];
					double * d_K_array[6]={Kxx, Kxy, Kxz, Kyy, Kyz, Kzz};
					for (int i = 0; i <6; i++)
					{
						checkCudaErrors(hipMemcpy( h_K_tmp,d_K_array[i], sizeof(double) * meshsize_padded,
																						hipMemcpyDeviceToHost));
						ofs.write(reinterpret_cast<char *>(h_K_tmp), sizeof(double) * meshsize_padded );
					}
					delete h_K_tmp;
				}
				catch(const std::exception &e)
				{cout<<"Fail to write cached tensor.\n";};
			}
			ofs.close();
		}
	}
;

#ifdef DEBUG_H_FIELD
	ovf_io::writetoOvfFile_device<double>(
						(output_path  / boost::filesystem::path("Kxx.ovf")).string(),
						ovfinfo,Kxx,Kyy,Kzz);
	ovf_io::writetoOvfFile_device<double>(
							(output_path  / boost::filesystem::path("Kxy.ovf")).string(),
							ovfinfo,Kxy,Kxz,Kyz);
#endif

#ifdef DISPLAY_VERBOSE_FFT
	double * KK[6] = { Kxx, Kxy, Kxz, Kyy, Kyz, Kzz };
	print_real<double>("K tensor ", KK, 6, nx_padded, ny_padded, nz_padded);
#endif
	//K tensor fft
	double2 * Kxx_fft_64, *Kxy_fft_64, *Kxz_fft_64, *Kyy_fft_64, *Kyz_fft_64,
			*Kzz_fft_64;
	FLOAT2 * Kxx_fft, *Kxy_fft, *Kxz_fft, *Kyy_fft, *Kyz_fft, *Kzz_fft;
	hipfftHandle plan_D2Z;
	cout<<"		Calculating FFT ...";
	hipfftPlan3d(&plan_D2Z, nx_padded, ny_padded, nz_padded, HIPFFT_D2Z);
	cout<<"plan done...";
	checkCudaErrors(
			hipMalloc((void ** )&Kxx_fft_64,
					sizeof(double2) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kxx_fft, sizeof(FLOAT2) * meshsize_padded));

	checkCudaErrors(
			hipMalloc((void ** )&Kxy_fft_64,
					sizeof(double2) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kxy_fft, sizeof(FLOAT2) * meshsize_padded));

	checkCudaErrors(
			hipMalloc((void ** )&Kxz_fft_64,
					sizeof(double2) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kxz_fft, sizeof(FLOAT2) * meshsize_padded));

	checkCudaErrors(
			hipMalloc((void ** )&Kyy_fft_64,
					sizeof(double2) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kyy_fft, sizeof(FLOAT2) * meshsize_padded));

	checkCudaErrors(
			hipMalloc((void ** )&Kyz_fft_64,
					sizeof(double2) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kyz_fft, sizeof(FLOAT2) * meshsize_padded));

	checkCudaErrors(
			hipMalloc((void ** )&Kzz_fft_64,
					sizeof(double2) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Kzz_fft, sizeof(FLOAT2) * meshsize_padded));

	//TODO: stream these things
	hipDeviceSynchronize();


	hipfftExecD2Z(plan_D2Z, Kxx, Kxx_fft_64);cout<<"1 done...";
	hipfftExecD2Z(plan_D2Z, Kxy, Kxy_fft_64);cout<<"2 done...";
	hipfftExecD2Z(plan_D2Z, Kxz, Kxz_fft_64);cout<<"3 done...";
	hipfftExecD2Z(plan_D2Z, Kyy, Kyy_fft_64);cout<<"4 done...";
	hipfftExecD2Z(plan_D2Z, Kyz, Kyz_fft_64);cout<<"5 done...";
	hipfftExecD2Z(plan_D2Z, Kzz, Kzz_fft_64);cout<<"6 done...\n";

	checkCudaErrors(hipFree(Kxx));
	checkCudaErrors(hipFree(Kxy));
	checkCudaErrors(hipFree(Kxz));
	checkCudaErrors(hipFree(Kyy));
	checkCudaErrors(hipFree(Kyz));
	checkCudaErrors(hipFree(Kzz));

	cout<<"		Copying ...";
	// copy K_fft_64 to K_fft (32)
	hipDeviceSynchronize();
	copyFFT64to32Kernel<<<dimGrid, dimBlock>>>(nx_padded, ny_padded, nz_padded,
			Kxx_fft_64, Kxy_fft_64, Kxz_fft_64, Kyy_fft_64, Kyz_fft_64,
			Kzz_fft_64, Kxx_fft, Kxy_fft, Kxz_fft, Kyy_fft, Kyz_fft, Kzz_fft);
	hipDeviceSynchronize();

	checkCudaErrors(hipFree(Kxx_fft_64));
	checkCudaErrors(hipFree(Kxy_fft_64));
	checkCudaErrors(hipFree(Kxz_fft_64));
	checkCudaErrors(hipFree(Kyy_fft_64));
	checkCudaErrors(hipFree(Kyz_fft_64));
	checkCudaErrors(hipFree(Kzz_fft_64));

	FLOAT2 *h_Kxx_fft = new FLOAT2[meshsize_padded]; //new FLOAT2[sizeof(FLOAT2) * meshsize_padded];

#ifdef DISPLAY_VERBOSE_FFT
	FLOAT2 * KK_fft[6] =
				{ Kxx_fft, Kxy_fft, Kxz_fft, Kyy_fft, Kyz_fft, Kzz_fft };
	print_complex<FLOAT2>("K tensor fft ", KK_fft, 6, nx_padded, ny_padded,
			nz_padded);
#endif
	delete h_Kxx_fft;
	cout<<"Done.\n";
	/* --------------------------  init Kxx fft-----------------------------*/

	/*=============================init Mx , Mx fft========================*/
//	float * Mx, * My, * Mz;
	FLOAT_ * Mx_padded, *My_padded, *Mz_padded;
	FLOAT_ * Mx_padded_output, *My_padded_output, *Mz_padded_output;

	FLOAT2 * Mx_padded_fft, *My_padded_fft, *Mz_padded_fft;

//	FLOAT_ * Hx_padded, *Hy_padded, *Hz_padded;
	FLOAT2 * Hx_padded_fft, *Hy_padded_fft, *Hz_padded_fft;

	checkCudaErrors(
			hipMalloc((void ** )&Mx_padded, sizeof(FLOAT_) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&My_padded, sizeof(FLOAT_) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Mz_padded, sizeof(FLOAT_) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Mx_padded_output, sizeof(FLOAT_) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&My_padded_output, sizeof(FLOAT_) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Mz_padded_output, sizeof(FLOAT_) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Mx_padded_fft, sizeof(FLOAT2) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&My_padded_fft, sizeof(FLOAT2) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Mz_padded_fft, sizeof(FLOAT2) * meshsize_padded));

	checkCudaErrors(
			hipMalloc((void ** )&Hx_padded_fft, sizeof(FLOAT2) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Hy_padded_fft, sizeof(FLOAT2) * meshsize_padded));
	checkCudaErrors(
			hipMalloc((void ** )&Hz_padded_fft, sizeof(FLOAT2) * meshsize_padded));
	/*----------------------------init Mx , Mx fft----------------------------------------*/

	//========================copy Mx to GPU================



	hipMemcpy(Mx_padded, h_Mx_padded, sizeof(FLOAT_) * meshsize_padded,
			hipMemcpyHostToDevice);
	hipMemcpy(My_padded, h_My_padded, sizeof(FLOAT_) * meshsize_padded,
			hipMemcpyHostToDevice);
	hipMemcpy(Mz_padded, h_Mz_padded, sizeof(FLOAT_) * meshsize_padded,
			hipMemcpyHostToDevice);
	hipMemcpy(Mx_padded_output, h_Mx_padded, sizeof(FLOAT_) * meshsize_padded,
			hipMemcpyHostToDevice);
	hipMemcpy(My_padded_output, h_My_padded, sizeof(FLOAT_) * meshsize_padded,
			hipMemcpyHostToDevice);
	hipMemcpy(Mz_padded_output, h_Mz_padded, sizeof(FLOAT_) * meshsize_padded,
			hipMemcpyHostToDevice);
	delete h_Mx_padded, h_My_padded, h_Mz_padded;
	//-----------------------copy Mx to GPU------------------

	//=====================init energy=================
	FLOAT_ * d_energy;
	checkCudaErrors(
				hipMalloc((void ** )&d_energy, sizeof(FLOAT_) * meshsize));


	/*=====================init  FFT plan=======================*/
	cout<<"Initializing FFT...";
#ifdef USE_DOULBE_IN_GPU
//	hipfftHandle plan_D2Z;
//	hipfftPlan3d(&plan_D2Z, nx_padded, ny_padded, nz_padded, HIPFFT_D2Z);
	hipfftHandle plan_Z2D;
	hipfftPlan3d(&plan_Z2D, nx_padded, ny_padded, nz_padded, HIPFFT_Z2D);
	hipfftHandle plan_Z2Z;
	hipfftPlan3d(&plan_Z2Z, nx_padded, ny_padded, nz_padded, HIPFFT_Z2Z);
#else
	hipfftHandle plan_R2C;
	hipfftHandle plan_C2R;
	hipfftPlan3d(&plan_R2C, nx_padded, ny_padded, nz_padded, HIPFFT_R2C);
	hipfftPlan3d(&plan_C2R, nx_padded, ny_padded, nz_padded, HIPFFT_C2R);
	hipfftHandle plan_C2C;
	hipfftPlan3d(&plan_C2C, nx_padded, ny_padded, nz_padded, HIPFFT_C2C);
#endif

	/*-----------------init  FFT plan-----------------------*/

#ifdef DISPLAY_VERBOSE_FFT
	FLOAT_ * handle_Mx_padded[3] = { Mx_padded, My_padded, Mz_padded };
	FLOAT2 * handle_M_fft_padded[3] = { Mx_padded_fft, My_padded_fft,
				Mz_padded_fft };
	FLOAT2 * handle_H_fft_padded[3] = { Hx_padded_fft, Hy_padded_fft,
				Hz_padded_fft };
	FLOAT_ * d_energy_p []= {d_energy};
#endif
	cout<<"Done.\n";


#ifdef DEBUG_H_FIELD
	FLOAT_ * d_debug_Hx, * d_debug_Hy, * d_debug_Hz;
	checkCudaErrors(
				hipMalloc((void ** )&d_debug_Hx, sizeof(FLOAT_) * meshsize_padded));
	checkCudaErrors(
					hipMalloc((void ** )&d_debug_Hy, sizeof(FLOAT_) * meshsize_padded));
	checkCudaErrors(
					hipMalloc((void ** )&d_debug_Hz, sizeof(FLOAT_) * meshsize_padded));
#endif

	// ==================================================================================================================
	// ==================================================================================================================
	// ==================================================================================================================
	// ======================================      where game begins   ==================================================
	// ==================================================================================================================
	// ==================================================================================================================
	// ==================================================================================================================
	cout<<"Start...\n";

	// ========= elapsed time====
//	std::chrono::time_point<std::chrono::system_clock> time_start, time_end;		//Method 'count' could not be resolved
//	std::chrono::duration<double> stage_elapsed_seconds;// = time_end - time_start;
//	time_start = std::chrono::system_clock::now();

	clock_t time_start;//, time_end;
	double stage_elapsed_seconds;
	//------------------------

	long output_file_count = 0;
	char filename_buff[200];
	while ( Temperature > Temperature_end)

	{
		printf("\n\nTemperature: %.2f K\n",Temperature);
		for (long stage=0; stage <circle_per_stage;stage++ )		//	FOR(stage,0,circle_per_stage)
		{
			if (CTRL_C_QUIT_FLAG) break;
			if (stage%terminal_output_period ==0 )
			{
				printf("Stage: %ld",stage);
				if (stage ==0)
				{
//					time_start = std::chrono::system_clock::now();
					time_start = clock();
					printf(" ");
				}
				else
				{
//					std::chrono::duration<double> vasdfa;// = time_end - time_start;
//					time_end = std::chrono::system_clock::now();
//					stage_elapsed_seconds = time_end - time_start;
					//printf(", elapsed time: %f s, speed: %f stages/s ",(stage_elapsed_seconds.count()), (stage_elapsed_seconds.count()/stage) );
					stage_elapsed_seconds = (double)(clock()-time_start)/CLOCKS_PER_SEC;
					printf(", elapsed time: %f s, speed: %f stages/s ",stage_elapsed_seconds, (stage/stage_elapsed_seconds) );

				}
			}
//			long energy_output_period = 1000;
//			long ms_output_period = 1000;
			if (stage % ms_output_period == 0) {
				{
//				double h_Mx[meshsize];
//				double h_My[meshsize];
//				double h_Mz[meshsize];
				double * h_Mx= new double[meshsize];
				double * h_My= new double[meshsize];
				double * h_Mz= new double[meshsize];

				h_Mx_padded = new FLOAT_[meshsize_padded];
				h_My_padded = new FLOAT_[meshsize_padded];
				h_Mz_padded = new FLOAT_[meshsize_padded];
				hipMemcpy(h_Mx_padded,Mx_padded,
						sizeof(FLOAT_) * meshsize_padded,
						hipMemcpyDeviceToHost);
				hipMemcpy( h_My_padded,My_padded,
						sizeof(FLOAT_) * meshsize_padded,
						hipMemcpyDeviceToHost);
				hipMemcpy(h_Mz_padded,Mz_padded,
						sizeof(FLOAT_) * meshsize_padded,
						hipMemcpyDeviceToHost);
				copyMxpaddedtoMx<FLOAT_,double>(
						h_Mx_padded,  h_My_padded, h_Mz_padded,  nx_padded,  ny_padded,  nz_padded,
						h_Mx,  h_My,  h_Mz,  nx,  ny,  nz);

				snprintf(filename_buff,200,"%06ld_%.2fK_%ld.ovf",output_file_count, Temperature, stage );
				std::string filename = filename_buff;
				filename =  (output_path / boost::filesystem::path(filename)).string();
				ovf_io::writetoOvfFile<double>( filename,ovfinfo,	h_Mx,  h_My,  h_Mz);

				delete h_Mx_padded, h_My_padded, h_Mz_padded;

				output_file_count++;
				delete h_Mx,h_My,h_Mz;
				}

			}

			if(cal_demag_flag)
			{
			#ifdef USE_DOULBE_IN_GPU
				hipfftExecD2Z(plan_D2Z, Mx_padded, Mx_padded_fft);
				hipfftExecD2Z(plan_D2Z, My_padded, My_padded_fft);
				hipfftExecD2Z(plan_D2Z, Mz_padded, Mz_padded_fft);
			#else
				hipfftExecR2C(plan_R2C, Mx_padded, Mx_padded_fft);
				hipfftExecR2C(plan_R2C, My_padded, My_padded_fft);
				hipfftExecR2C(plan_R2C, Mz_padded, Mz_padded_fft);
			#endif
				hipDeviceSynchronize();
				calHexchangeKernel<<<dimGrid, dimBlock>>>(nx_padded, ny_padded, nz_padded,
						Mx_padded_fft, My_padded_fft, Mz_padded_fft, Kxx_fft, Kxy_fft,
						Kxz_fft, Kyy_fft, Kyz_fft, Kzz_fft, Hx_padded_fft, Hy_padded_fft,
						Hz_padded_fft);
				hipDeviceSynchronize();

			#ifdef DISPLAY_VERBOSE_FFT
				print_real<FLOAT_>("M padded ", handle_Mx_padded, 3, nx_padded, ny_padded,
						nz_padded, 0);
				print_complex<FLOAT2>("M padded FFT ", handle_M_fft_padded, 3, nx_padded,
						ny_padded, nz_padded, 0);
				print_complex<FLOAT2>("H_fft_ (M_fft*K_fft) ", handle_H_fft_padded, 3, nx_padded,
						ny_padded, nz_padded, 0);
			#endif


			#ifdef DISPLAY_VERBOSE_FFT
				print_complex<FLOAT2>("H_fft_INVERSE ", handle_H_fft_padded, 3, nx_padded,
						ny_padded, nz_padded, 0);
			#endif
			}

			if (stage%energy_output_period == 0)
			{
				// =========================== cal exchange field energy=========
						hipDeviceSynchronize();
//						effectiveHKernel<<<dimGrid_nopadding, dimBlock_nopadding>>>( ms, factor_H_exch, Dind,
//									Ku1, anisUx, anisUy, anisUz,
//									Bextx, Bexty, Bextz,
//									nx, ny, nz,
//									nx_padded, ny_padded, nz_padded, dx, dy, dz,
//									pbc_x, pbc_y, pbc_z,
//									Mx_padded, My_padded, Mz_padded,
//									Hx_padded_fft, Hy_padded_fft, Hz_padded_fft,
//									d_energy, cal_demag_flag);
						calEnergyKernel<<<dimGrid_nopadding, dimBlock_nopadding>>>( ms,  mu0, factor_H_exch,  Aex,  Dind,
											 Ku1,  anisUx,  anisUy,  anisUz,
											 Bextx,  Bexty,  Bextz,
											 nx,  ny,  nz,
											 nx_padded,  ny_padded,  nz_padded,  dx,  dy,
											 dz,  pbc_x,  pbc_y,  pbc_z,
											  Mx_padded, My_padded, Mz_padded,
											 Hx_padded_fft, Hy_padded_fft, Hz_padded_fft,
											 d_energy,  cal_demag_flag) ;
						hipDeviceSynchronize();
//						print_real<FLOAT_>("M padded ", handle_Mx_padded, 3, nx_padded, ny_padded,
//											nz_padded, 0);
						#ifdef DISPLAY_VERBOSE_FFT
//							print_complex<FLOAT2>("H demag + i * H exch ", handle_H_fft_padded, 3,
//									nx_padded, ny_padded, nz_padded, 0);
						// DISPALY_H_demag_H_demag_
							print_complex_real_nopadding<FLOAT2>("H demag ", handle_H_fft_padded, 3,
															nx_padded, ny_padded, nz_padded,
															nx,ny,nz,0);
							print_complex_imag_nopadding<FLOAT2>("H exch ", handle_H_fft_padded, 3,
															nx_padded, ny_padded, nz_padded,
															nx,ny,nz,0);
							print_real<FLOAT_>("Energy ", d_energy_p, 1, nx, ny, nz,0);
						#endif
//							FLOAT_ * handle_Mx_padded[3] = { Mx_padded, My_padded, Mz_padded };
//							print_real<FLOAT_>("M padded ", handle_Mx_padded, 3, nx_padded, ny_padded,
//												nz_padded, 0);
//							FLOAT_ * d_energy_p []= {d_energy};
//							print_real<FLOAT_>("Energy ", d_energy_p, 1, nx, ny, nz,0);
							//cout<<" thrust reduce "<<endl;
							thrust::device_ptr<FLOAT_> d_sum_ptr = thrust::device_pointer_cast(d_energy);
//							FLOAT_ tot_energy = thrust::reduce(d_sum_ptr, d_sum_ptr + meshsize);
							FLOAT_ tot_energy = thrust::reduce(d_sum_ptr, d_sum_ptr + meshsize)*dx*dy*dz;
							printf("\nEnergy:  %g\n", tot_energy);

							{
								FILE *fp;

								fp = fopen(energy_filename_string.c_str(), "a");
								fprintf(fp,"%f %ld %g\n",Temperature,stage,tot_energy);
								fclose(fp);
							}



			}

			//====================================main Monte ======================================


#ifdef DEBUG_H_FIELD
			FLOAT_ *  h_debug_H[3] = {d_debug_Hx, d_debug_Hy, d_debug_Hz};
			cout<<"###########################DEBUG_B_demag##############\n";
			mcKernel_debug_demag<<<dimGrid_nopadding, dimBlock_nopadding>>>(
								 ms,  mu0,  Aex,  Dind,
								 Ku1,  anisUx,  anisUy,  anisUz,
								 Bextx, Bexty, Bextz,
								 (FLOAT_)(CONST_K_BOLTZMANN * Temperature),
								 nx,  ny,  nz,  nx_padded,  ny_padded,
								 nz_padded,  meshsize,  dx,  dy,  dz,  pbc_x,  pbc_y,  pbc_z,
								  Kxx_mirror,  Kyy_mirror,   Kzz_mirror,
								 Mx_padded, My_padded, Mz_padded,
								 Mx_padded_output, My_padded_output, Mz_padded_output,
								 Hx_padded_fft, Hy_padded_fft, Hz_padded_fft,
								 dRandPool,  dRandPool_reject,  randPoolCount, cal_demag_flag,
								 d_debug_Hx, d_debug_Hy, d_debug_Hz);
//			print_real<FLOAT_>("B demag", h_debug_H, 3,
//					nx_padded, ny_padded, nz_padded, 0);
			print_real_along_col<FLOAT_>("B demag", d_debug_Hx,d_debug_Hy,d_debug_Hz, 3,
								nx_padded, ny_padded, nz_padded, 0);
			ovf_io::writetoOvfFile_device<FLOAT_>(
					(output_path  / boost::filesystem::path("B_demag.ovf")).string(),
					ovfinfo,	d_debug_Hx,d_debug_Hy,d_debug_Hz);
			//cout<<"###########################DEBUG_H_FIELD##############\n";
			cout<<"###########################DEBUG_B_anis##############\n";
			mcKernel_debug_anis<<<dimGrid_nopadding, dimBlock_nopadding>>>(
								 ms,  mu0,  Aex,  Dind,
								 Ku1,  anisUx,  anisUy,  anisUz,
								 Bextx, Bexty, Bextz,
								 (FLOAT_)(CONST_K_BOLTZMANN * Temperature),
								 nx,  ny,  nz,  nx_padded,  ny_padded,
								 nz_padded,  meshsize,  dx,  dy,  dz,  pbc_x,  pbc_y,  pbc_z,
								  Kxx_mirror,  Kyy_mirror,   Kzz_mirror,
								 Mx_padded, My_padded, Mz_padded,
								 Mx_padded_output, My_padded_output, Mz_padded_output,
								 Hx_padded_fft, Hy_padded_fft, Hz_padded_fft,
								 dRandPool,  dRandPool_reject,  randPoolCount, cal_demag_flag,
								 d_debug_Hx, d_debug_Hy, d_debug_Hz);
//			print_real<FLOAT_>("H anis", h_debug_H, 3,
//					nx_padded, ny_padded, nz_padded, 0);
			print_real_along_col<FLOAT_>("B anis", d_debug_Hx,d_debug_Hy,d_debug_Hz, 3,
											nx_padded, ny_padded, nz_padded, 0);
			ovf_io::writetoOvfFile_device<FLOAT_>(
					(output_path  / boost::filesystem::path("B_anis.ovf")).string(),
					ovfinfo,d_debug_Hx,d_debug_Hy,d_debug_Hz);
			cout<<"###########################DEBUG_B_exch##############\n";
			mcKernel_debug_exch<<<dimGrid_nopadding, dimBlock_nopadding>>>(
											 ms,  mu0,  Aex,  Dind,
											 Ku1,  anisUx,  anisUy,  anisUz,
											 Bextx, Bexty, Bextz,
											 (FLOAT_)(CONST_K_BOLTZMANN * Temperature),
											 nx,  ny,  nz,  nx_padded,  ny_padded,
											 nz_padded,  meshsize,  dx,  dy,  dz,  pbc_x,  pbc_y,  pbc_z,
											  Kxx_mirror,  Kyy_mirror,   Kzz_mirror,
											 Mx_padded, My_padded, Mz_padded,
											 Mx_padded_output, My_padded_output, Mz_padded_output,
											 Hx_padded_fft, Hy_padded_fft, Hz_padded_fft,
											 dRandPool,  dRandPool_reject,  randPoolCount, cal_demag_flag,
											 d_debug_Hx, d_debug_Hy, d_debug_Hz);
//			print_real<FLOAT_>("B_exch", h_debug_H, 3,
//					nx_padded, ny_padded, nz_padded, 0);
			print_real_along_col<FLOAT_>("B exch", d_debug_Hx,d_debug_Hy,d_debug_Hz, 3,
														nx_padded, ny_padded, nz_padded, 0);
			ovf_io::writetoOvfFile_device<FLOAT_>(
					(output_path  / boost::filesystem::path("B_exch.ovf")).string(),
					ovfinfo,d_debug_Hx,d_debug_Hy,d_debug_Hz);
			cout<<"###########################DEBUG_B_effective##############\n";
			mcKernel_debug_total<<<dimGrid_nopadding, dimBlock_nopadding>>>(
											 ms,  mu0,  Aex,  Dind,
											 Ku1,  anisUx,  anisUy,  anisUz,
											 Bextx, Bexty, Bextz,
											 (FLOAT_)(CONST_K_BOLTZMANN * Temperature),
											 nx,  ny,  nz,  nx_padded,  ny_padded,
											 nz_padded,  meshsize,  dx,  dy,  dz,  pbc_x,  pbc_y,  pbc_z,
											  Kxx_mirror,  Kyy_mirror,   Kzz_mirror,
											 Mx_padded, My_padded, Mz_padded,
											 Mx_padded_output, My_padded_output, Mz_padded_output,
											 Hx_padded_fft, Hy_padded_fft, Hz_padded_fft,
											 dRandPool,  dRandPool_reject,  randPoolCount, cal_demag_flag,
											 d_debug_Hx, d_debug_Hy, d_debug_Hz);
			print_real_along_col<FLOAT_>("B_effective", d_debug_Hx,d_debug_Hy,d_debug_Hz, 3,
					nx_padded, ny_padded, nz_padded, 0);
			ovf_io::writetoOvfFile_device<FLOAT_>(
					(output_path  / boost::filesystem::path("B_effective.ovf")).string(),
					ovfinfo,d_debug_Hx,d_debug_Hy,d_debug_Hz);
			cout<<"###########################DEBUG_E_density##############\n";
			calEnergyKernel<<<dimGrid_nopadding, dimBlock_nopadding>>>( ms,  mu0, factor_H_exch,  Aex,  Dind,
					 Ku1,  anisUx,  anisUy,  anisUz,
					 Bextx,  Bexty,  Bextz,
					 nx,  ny,  nz,
					 nx_padded,  ny_padded,  nz_padded,  dx,  dy,
					 dz,  pbc_x,  pbc_y,  pbc_z,
					  Mx_padded, My_padded, Mz_padded,
					 Hx_padded_fft, Hy_padded_fft, Hz_padded_fft,
					 d_energy,  cal_demag_flag) ;
			print_real_along_col<FLOAT_>("E_density", d_energy,d_energy,d_energy, 3,
					nx_padded, ny_padded, nz_padded, 0);
			ovf_io::writetoOvfFile_device<FLOAT_>(
					(output_path  / boost::filesystem::path("E_density.ovf")).string(),
					ovfinfo,d_energy,d_energy,d_energy);
			cout<<"###########################DEBUG_mc_E_density##############\n";
			mcKernel2<<<dimGrid_nopadding, dimBlock_nopadding>>>(
								 ms,  mu0,  Aex,  Dind,
								 Ku1,  anisUx,  anisUy,  anisUz,
								 Bextx, Bexty, Bextz,
								 (FLOAT_)(CONST_K_BOLTZMANN * Temperature),
								 nx,  ny,  nz,  nx_padded,  ny_padded,
								 nz_padded,  meshsize,  dx,  dy,  dz,  pbc_x,  pbc_y,  pbc_z,
								 Kxx_mirror, Kxy_mirror, Kxz_mirror,
								 Kyy_mirror, Kyz_mirror, Kzz_mirror,
								 Mx_padded, My_padded, Mz_padded,
								 Mx_padded_output, My_padded_output, Mz_padded_output,
								 Hx_padded_fft, Hy_padded_fft, Hz_padded_fft,
								 dRandPool,  dRandPool_reject,  randPoolCount, cal_demag_flag,
								 d_energy);
			print_real_along_col<FLOAT_>("mc_E_density", d_energy,d_energy,d_energy, 3,
					nx_padded, ny_padded, nz_padded, 0);
			ovf_io::writetoOvfFile_device<FLOAT_>(
					(output_path  / boost::filesystem::path("mc_E_density.ovf")).string(),
					ovfinfo,d_energy,d_energy,d_energy);
			cout<<"###########################DEBUG_mc_E_density##############\n";
#endif

//			mcKernel<<<dimGrid_nopadding, dimBlock_nopadding>>>(
//					 ms,  mu0,  Aex,  Dind,
//					 Ku1,  anisUx,  anisUy,  anisUz,
//					 Bextx, Bexty, Bextz,
//					 (FLOAT_)(CONST_K_BOLTZMANN * Temperature),
//					 nx,  ny,  nz,  nx_padded,  ny_padded,
//					 nz_padded,  meshsize,  dx,  dy,  dz,  pbc_x,  pbc_y,  pbc_z,
//					  Kxx_mirror,  Kyy_mirror,   Kzz_mirror,
//					 Mx_padded, My_padded, Mz_padded,
//					 Mx_padded_output, My_padded_output, Mz_padded_output,
//					 Hx_padded_fft, Hy_padded_fft, Hz_padded_fft,
//					 dRandPool,  dRandPool_reject,  randPoolCount, cal_demag_flag);
			// TODO: CONST_K_BOLTZMANN * Temperature * 3
			mcKernel2<<<dimGrid_nopadding, dimBlock_nopadding>>>(
											 ms,  mu0,  Aex,  Dind,
											 Ku1,  anisUx,  anisUy,  anisUz,
											 Bextx, Bexty, Bextz,
											 (FLOAT_)(CONST_K_BOLTZMANN * Temperature),
											 nx,  ny,  nz,  nx_padded,  ny_padded,
											 nz_padded,  meshsize,  dx,  dy,  dz,  pbc_x,  pbc_y,  pbc_z,
											 Kxx_mirror, Kxy_mirror, Kxz_mirror,
											 Kyy_mirror, Kyz_mirror, Kzz_mirror,
											 Mx_padded, My_padded, Mz_padded,
											 Mx_padded_output, My_padded_output, Mz_padded_output,
											 Hx_padded_fft, Hy_padded_fft, Hz_padded_fft,
											 dRandPool,  dRandPool_reject,  randPoolCount, cal_demag_flag,
											 d_energy);
			hipDeviceSynchronize();
			randPoolCount++;

			swap(Mx_padded,Mx_padded_output);
			swap(My_padded,My_padded_output);
			swap(Mz_padded,Mz_padded_output);

			if (CTRL_C_QUIT_FLAG) break;
		}
		//stage end

		if (CTRL_C_QUIT_FLAG) break;
	// ==================================  update Temperature ================

		if (Temperature_use_exp)
		{
			Temperature =  Temperature * Temperature_step;
		}
		else
		{
			Temperature =  Temperature - Temperature_step;
		}


	}

	//----------------------------------------------------------------------------------------------------------------
	//----------------------------------------------------------------------------------------------------------------
	//----------------------------------------------------------------------------------------------------------------
	//--------------------------------------------      running end   ------------------------------------------
	//----------------------------------------------------------------------------------------------------------------
	//----------------------------------------------------------------------------------------------------------------
	//----------------------------------------------------------------------------------------------------------------

//	int x, y, z;
//	int shifted_x, shifted_y, shifted_z;
//	shifted_x = (x + nx_padded / 2 - 1) % nx_padded;
//	shifted_y = (x + ny_padded / 2 - 1) % ny_padded;
//	shifted_z = (x + nz_padded / 2 - 1) % nz_padded;

//	cout << "----------------" << endl;
	//!!!   Hx_padded = Hx_padded/meshsize_padded;
//	hipMemcpy(h_float_padded, Hx_padded, sizeof(FLOAT_) * meshsize_padded,
//			hipMemcpyDeviceToHost);
//	cout << "--------Hx_padded--------" << endl;
//	for (int kk = 0; kk < nz_padded; kk++) {
//		cout << "nz = " << kk << endl;
//		for (int ii = 0; ii < nx_padded; ii++) {
//			for (int jj = 0; jj < ny_padded; jj++)
////		 				cout << h_Hx[kk + jj * nz_padded + ii * nz_padded * ny_padded]/meshsize_padded
////		 						<< " ";
//				printf("%.4f ",
//						h_float_padded[kk + jj * nz_padded
//								+ ii * nz_padded * ny_padded]
//								/ meshsize_padded);
//			cout << endl;
//		}
//	}
//	cout << "----------------" << endl;

//	cout << "----------------" << endl;
	//!!!   Hx_padded = Hx_padded/meshsize_padded;
//	hipMemcpy(h_float_padded, Hx_padded, sizeof(FLOAT_) * meshsize_padded,
//			hipMemcpyDeviceToHost);
//	cout << "--------Hx_padded shifted--------" << endl;
//	for (int kk = 0; kk < nz_padded; kk++) {
//		cout << "nz = " << kk << endl;
//		for (int ii = 0; ii < nx_padded; ii++) {
//			for (int jj = 0; jj < ny_padded; jj++) {
//				int x = (ii + nx_padded / 2 - 1) % nx_padded;
//				int y = (jj + ny_padded / 2 - 1) % ny_padded;
//				int z = (kk + nz_padded / 2 - 1) % nz_padded;
//				printf("%.4f ",
//						h_float_padded[z + y * nz_padded
//								+ x * nz_padded * ny_padded] / meshsize_padded);
//			}
//			cout << endl;
//		}
//	}
//	cout << "----------------" << endl;


	//=========================  clean field==========================


	checkCudaErrors(hipFree(Kxx_fft));
	checkCudaErrors(hipFree(Kxy_fft));
	checkCudaErrors(hipFree(Kxz_fft));
	checkCudaErrors(hipFree(Kyy_fft));
	checkCudaErrors(hipFree(Kyz_fft));
	checkCudaErrors(hipFree(Kzz_fft));

//	float * Mx_padded, * My_padded, * Mz_padded;
//	float2 * Mx_padded_fft, * My_padded_fft,* Mz_padded_fft;
	checkCudaErrors(hipFree(Mx_padded));
	checkCudaErrors(hipFree(My_padded));
	checkCudaErrors(hipFree(Mz_padded));
	checkCudaErrors(hipFree(Mx_padded_output));
	checkCudaErrors(hipFree(My_padded_output));
	checkCudaErrors(hipFree(Mz_padded_output));
	checkCudaErrors(hipFree(Mx_padded_fft));
	checkCudaErrors(hipFree(My_padded_fft));
	checkCudaErrors(hipFree(Mz_padded_fft));
//	float * Hx_padded, * Hy_padded, * Hz_padded;
//	float2 * Hx_padded_fft, * Hy_padded_fft,* Hz_padded_fft;

//	checkCudaErrors(hipFree(Hx_padded));
//	checkCudaErrors(hipFree(Hy_padded));
//	checkCudaErrors(hipFree(Hz_padded));

	checkCudaErrors(hipFree(Hx_padded_fft));
	checkCudaErrors(hipFree(Hy_padded_fft));
	checkCudaErrors(hipFree(Hz_padded_fft));

	checkCudaErrors(hipFree(dRandPool));
	CURAND_CALL(hiprandDestroyGenerator(randgen));

	checkCudaErrors(hipFree(d_energy));


	hipfftDestroy(plan_D2Z);

//	hipfftDestroy(plan_Z2Z);
#ifdef USE_DOULBE_IN_GPU
	hipfftDestroy(plan_Z2D);
	hipfftDestroy(plan_Z2Z);
#else
	hipfftDestroy(plan_R2C);
	hipfftDestroy(plan_C2R);
	hipfftDestroy(plan_C2C);
#endif


#ifdef DEBUG_H_FIELD
	checkCudaErrors(hipFree(d_debug_Hx));
	checkCudaErrors(hipFree(d_debug_Hy));
	checkCudaErrors(hipFree(d_debug_Hz));
#endif




//	hipFloatComplex a,b;
//	a.x=1;a.y=1;
//	b.x =3;b.y=-1;
//	hipDoubleComplex c;
//	c = hipCmul(a,b);
//	cout<<c.x<<endl;
//	cout<<c.y<<endl;
	cout<<"~~~~~~~~~~~~~~END~~~~~~~~~~~~~~~~~~~~~~~~~ \n";
	return 0;
}
