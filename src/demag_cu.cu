#include "hip/hip_runtime.h"
/*
 * demag_cu.h
 *
 *  Created on: 2 Nov, 2016
 *      Author: cg
 */
#include "demag_cu.h"
#include <hip/hip_complex.h>

#define ComplexXreal(a,b) ((a).x * (b).x - (a).y * (b).y)
#define ComplexXimag(a,b) ((a).x * (b).y + (a).y * (b).x)
#define ComplexSet(a,b)  (a).x = (b).x ; (a).y = (b).y;

#define ABS(X)   (((X)>0)?(X):(-X))

__inline__ __device__ double f(double x, double y, double z) {
	x = ABS(x);
	y = ABS(y);
	z = ABS(z);

	return +y / 2.0 * (z * z - x * x)
			* asinh(y / (sqrt(x * x + z * z) + EPSILON))
			+ z / 2.0 * (y * y - x * x)
					* asinh(z / (sqrt(x * x + y * y) + EPSILON))
			- x * y * z
					* atan(y * z / (x * sqrt(x * x + y * y + z * z) + EPSILON))
			+ 1.0 / 6.0 * (2 * x * x - y * y - z * z)
					* sqrt(x * x + y * y + z * z);
}
__inline__ __device__ double g(double x, double y, double z) {
	z = ABS(z);
	return +x * y * z * asinh(z / (sqrt(x * x + y * y) + EPSILON))
			+ y / 6.0 * (3.0 * z * z - y * y)
					* asinh(x / (sqrt(y * y + z * z) + EPSILON))
			+ x / 6.0 * (3.0 * z * z - x * x)
					* asinh(y / (sqrt(x * x + z * z) + EPSILON))
			- z * z * z / 6.0
					* atan(x * y / (z * sqrt(x * x + y * y + z * z) + EPSILON))
			- z * y * y / 2.0
					* atan(x * z / (y * sqrt(x * x + y * y + z * z) + EPSILON))
			- z * x * x / 2.0
					* atan(y * z / (x * sqrt(x * x + y * y + z * z) + EPSILON))
			- x * y * sqrt(x * x + y * y + z * z) / 3.0;
}

__global__ void ktensorKernel2_step0_init(int nx_padded, int ny_padded, int nz_padded,
		double * Kxx, double * Kxy, double * Kxz, double * Kyy, double * Kyz,
		double * Kzz) {

	// TODO: ignore K(0,0) term, to remove H acting on (0,0)
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	int threadId = z + y * nz_padded + x * nz_padded * ny_padded;
	if (x >= nx_padded || y >= ny_padded || z >= nz_padded)
		return;

	Kxx[threadId] = 0;
	Kxy[threadId] = 0;
	Kxz[threadId] = 0;
	Kyy[threadId] = 0;
	Kyz[threadId] = 0;
	Kzz[threadId] = 0;

}


__global__ void calHexchangeKernel(int nx_padded, int ny_padded, int nz_padded,
		FLOAT2 * Mx_padded_fft, FLOAT2 * My_padded_fft, FLOAT2 * Mz_padded_fft,
		FLOAT2 * Kxx_fft, FLOAT2 * Kxy_fft, FLOAT2 * Kxz_fft, FLOAT2 * Kyy_fft,
		FLOAT2 * Kyz_fft, FLOAT2 * Kzz_fft, FLOAT2 * Hx_padded_fft,
		FLOAT2 * Hy_padded_fft, FLOAT2 * Hz_padded_fft) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	int threadId = z + y * nz_padded + x * ny_padded * nz_padded;
	if (x >= nx_padded || y >= ny_padded || z >= nz_padded)
		return;

	unsigned int  normal_factor = nx_padded * ny_padded * nz_padded;

	FLOAT2 Mx, My, Mz;
	FLOAT2 Kxx, Kxy, Kxz, Kyy, Kyz, Kzz;

	ComplexSet(Mx, Mx_padded_fft[threadId]);
	ComplexSet(My, My_padded_fft[threadId]);
	ComplexSet(Mz, Mz_padded_fft[threadId]);

	ComplexSet(Kxx, Kxx_fft[threadId]);
	ComplexSet(Kxy, Kxy_fft[threadId]);
	ComplexSet(Kxz, Kxz_fft[threadId]);
	ComplexSet(Kyy, Kyy_fft[threadId]);
	ComplexSet(Kyz, Kyz_fft[threadId]);
	ComplexSet(Kzz, Kzz_fft[threadId]);


	Hx_padded_fft[threadId].x = (ComplexXreal(Mx, Kxx) + ComplexXreal(My,Kxy)
	+ ComplexXreal(Mz,Kxz))/normal_factor;
	Hx_padded_fft[threadId].y = (ComplexXimag(Mx, Kxx) + ComplexXimag(My,Kxy)
	+ ComplexXimag(Mz,Kxz))/normal_factor;

	Hy_padded_fft[threadId].x = (ComplexXreal(Mx, Kxy) + ComplexXreal(My,Kyy)
	+ ComplexXreal(Mz,Kyz))/normal_factor;
	Hy_padded_fft[threadId].y = (ComplexXimag(Mx, Kxy) + ComplexXimag(My,Kyy)
	+ ComplexXimag(Mz,Kyz))/normal_factor;

	Hz_padded_fft[threadId].x = (ComplexXreal(Mx, Kxz) + ComplexXreal(My,Kyz)
	+ ComplexXreal(Mz,Kzz))/normal_factor;
	Hz_padded_fft[threadId].y = (ComplexXimag(Mx, Kxz) + ComplexXimag(My,Kyz)
	+ ComplexXimag(Mz,Kzz))/normal_factor;




}


void ktensorcpu::ktensorCPU (int nx_padded, int ny_padded, int nz_padded,
		double dx, double dy, double dz, int pbc_x, int pbc_y, int pbc_z,
		double * Kxx_d, double * Kxy_d, double * Kxz_d, double * Kyy_d, double * Kyz_d,
		double * Kzz_d, bool & CTRL_C_QUIT_FLAG)
{
		using namespace std;


	    long meshsize_padded = nx_padded * ny_padded * nz_padded;

	    printf("      ");

	    double * Kxx = new double[sizeof(double)*meshsize_padded];
	    double * Kxy = new double[sizeof(double)*meshsize_padded];
	    double * Kyy = new double[sizeof(double)*meshsize_padded];
	    double * Kxz = new double[sizeof(double)*meshsize_padded];
	    double * Kyz = new double[sizeof(double)*meshsize_padded];
	    double * Kzz = new double[sizeof(double)*meshsize_padded];

	    boost::progress_display show_progress( meshsize_padded );

	    FOR(x,0,nx_padded)
	    {
	        FOR(y,0,ny_padded)
	        {
	            FOR(z,0,nz_padded)
	            {
	                int threadId = z + y * nz_padded + x * nz_padded * ny_padded;

	                int idx_x, idx_y, idx_z;
	                idx_x = (x + (nx_padded+1)/2-1  )%( nx_padded ) - (nx_padded+1)/2 + 1;
	                idx_y = (y + (ny_padded+1)/2-1  )%( ny_padded ) - (ny_padded+1)/2 + 1;
	                idx_z = (z + (nz_padded+1)/2-1  )%( nz_padded ) - (nz_padded+1)/2 + 1;

	                double tmp;
	                double kxx, kxy, kxz, kyy, kyz, kzz;
	                kxx=0;kxy=0;kxz=0;kyy=0;kyz=0;kzz=0;

	                for (int pbc_idx_x = -pbc_x; pbc_idx_x <= pbc_x; pbc_idx_x++)
	                {
	                    for (int pbc_idx_y = -pbc_y; pbc_idx_y <= pbc_y; pbc_idx_y++)
	                    {
	                        for (int pbc_idx_z = -pbc_z; pbc_idx_z <= pbc_z; pbc_idx_z++)
	                        {

	                            int ix = idx_x + pbc_idx_x*nx_padded;
	                            int iy = idx_y + pbc_idx_y*ny_padded;
	                            int iz = idx_z + pbc_idx_z*nz_padded;

	                            int iix,iiy,iiz;
	                            bool sign;
	                            FOR01(ikx)
	                            {
	                                FOR01(iky)
	                                {
	                                    FOR01(ikz)
	                                    {
	                                        FOR01(ilx)
	                                        {
	                                            FOR01(ily)
	                                            {
	                                                FOR01(ilz)
	                                                {

	                                                    iix = ix+ikx-ilx;
	                                                    iiy = iy+iky-ily;
	                                                    iiz = iz+ikz-ilz;


	                                                    sign = ((ikx + iky + ikz + ilx + ily + ilz)%2 == 0);

	                                                    tmp = f(iix*dx,iiy*dy,iiz*dz);
	                                                    //cout<<"f :"<<tmp<<endl;
	                                                    kxx += (sign? tmp: -tmp);

	                                                    tmp = g(iix*dx,iiy*dy,iiz*dz);	//xy
	                                                    kxy += (sign? tmp: -tmp);

	                                                    tmp = g(iix*dx,iiz*dz,iiy*dy);	//xz
	                                                    kxz += (sign? tmp: -tmp);

	                                                    tmp = f(iiy*dy,iiz*dz,iix*dx);
	                                                    kyy += (sign? tmp: -tmp);

	                                                    tmp = g(iiy*dy,iiz*dz,iix*dx);	//yz
	                                                    kyz += (sign? tmp: -tmp);

	                                                    tmp = f(iiz*dz,iix*dx,iiy*dy);
	                                                    kzz += (sign? tmp: -tmp);
	                                                }
	                                            }
	                                        }
	                                    }
	                                }
	                            }



	                            //end PBC
	                        }
	                    }
	                }
	                tmp = -CONST_PI*4.0*dx*dy*dz;

	                Kxx[threadId] = kxx / tmp;
	                Kxy[threadId] = kxy / tmp;
	                Kxz[threadId] = kxz / tmp;
	                Kyy[threadId] = kyy / tmp;
	                Kyz[threadId] = kyz / tmp;
	                Kzz[threadId] = kzz / tmp;

	                ++show_progress;
	                if (CTRL_C_QUIT_FLAG) break;
	            }
	            if (CTRL_C_QUIT_FLAG) break;
	        }
	        if (CTRL_C_QUIT_FLAG) break;
	    }
	    double * KK[6]={Kxx,Kxy,Kxz,Kyy,Kyz,Kzz};

	    double * d_K_array[6]={Kxx_d, Kxy_d, Kxz_d, Kyy_d, Kyz_d, Kzz_d};
		for (int i = 0; i <6; i++)
		{
			checkCudaErrors(hipMemcpy(d_K_array[i], KK[i], sizeof(double) * meshsize_padded,
											hipMemcpyHostToDevice));
		}

	    delete Kxx,Kxy,Kxz,Kyy,Kyz,Kzz;

}


